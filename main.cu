#include "hip/hip_runtime.h"
%%cu
#include ""
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N 2000
#define DIV 100

// Выполняется на GPU
__global__ void mult_matrix(int* matrix1, int* matrix2, int* result, int i) {
	int k = blockIdx.x * (N / DIV) + threadIdx.x;
	int j = blockIdx.y * (N / DIV) + threadIdx.y;
	result[k * N + j] += matrix1[k * N + i] * matrix2[i * N + j];
}

__host__ void insert_matrix(int* matrix){
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j)
			matrix[i * N + j] = rand() % 10;
	}
}

int main() {

	printf("N = %d, DIV = %d \n", N, DIV);

	//заполнение матриц
	int *matrix1,
		*matrix2,
		*result;

	matrix1 = new int[N * N];
	matrix2 = new int[N * N];
	result = new int[N * N];

	insert_matrix(matrix1);
	insert_matrix(matrix2);
	memset(result, 0, N*N);

	int* dev_matrix1, *dev_matrix2, *dev_result;
	hipError_t cudaStatus;

	// Выделение памяти на видеокарте
	hipMalloc((void**)&dev_matrix1, N * N * sizeof(int));
	hipMalloc((void**)&dev_matrix2, N * N * sizeof(int));
	hipMalloc((void**)&dev_result, N * N * sizeof(int));

	hipError_t error;
	// Копирование матриц из оперативной памяти в память видеокарты
	error = hipMemcpy(dev_matrix1, matrix1, N * N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
		printf("%s\n", hipGetErrorString(error));

	error = hipMemcpy(dev_matrix2, matrix2, N * N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
		printf("%s\n", hipGetErrorString(error));

	error = hipMemcpy(dev_result, result, N * N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
		printf("%s\n", hipGetErrorString(error));

	dim3 grid(DIV, DIV);
	dim3 blocks(N / DIV, N / DIV);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	//запускаем алгоритм
	for (int i = 0; i < N; ++i) 
		mult_matrix << <grid, blocks >> > (dev_matrix1, dev_matrix2, dev_result, i);

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	//записываем время работы
	float time = 0;
	hipEventElapsedTime(&time, start, stop);

	error = hipGetLastError();
	if (error != hipSuccess)
		printf("%s\n", hipGetErrorString(error));


	hipDeviceSynchronize();

	error = hipMemcpy(matrix1, dev_matrix1, N * N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
		printf("%s\n", hipGetErrorString(error));

	error = hipMemcpy(matrix2, dev_matrix2, N * N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
		printf("%s\n", hipGetErrorString(error));

	error = hipMemcpy(result, dev_result, N * N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
		printf("%s\n", hipGetErrorString(error));

	printf("\nTIME: %fs \n", time/1000);

	delete matrix1;
	delete matrix2;
	delete result;
	hipFree(dev_matrix1);
	hipFree(dev_matrix2);
	hipFree(dev_result);
	return 0;
}